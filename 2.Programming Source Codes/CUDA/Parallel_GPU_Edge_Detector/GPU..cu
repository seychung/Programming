#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/timeb.h>

__global__ void smooth(float *smooth_fid,float *fid,float *smooth_h,int M)
{
const int smooth_L = 7;
const int h_size = 15;
__shared__ float h[h_size][h_size];
// Fill the filter
for(int i = 0; i < 15; i++) 
  for(int j = 0; j < 15; j++) 
    h[i][j] = smooth_h[i* h_size + j];

const int pixel_address = threadIdx.x + (blockIdx.x *blockDim.x) +(threadIdx.y *M ) + (blockIdx.y * blockDim.y) * M;
float sum = 0;
float value = 0;

for (int i = -smooth_L; i <= smooth_L; i++)	
  for (int j = -smooth_L; j <= smooth_L; j++)	
    {
      // check row 
      if (blockIdx.x == 0 && (threadIdx.x + i) < 0)	// left boarder
        value = 0;
      else if ( blockIdx.x == (gridDim.x - 1) && (threadIdx.x + i) > blockDim.x-1 )	// right boarder
        value = 0;
      else 
      { 
        // check column  
        if (blockIdx.y == 0 && (threadIdx.y + j) < 0)	// top boarder
          value = 0;
        else if ( blockIdx.y == (gridDim.y - 1) && (threadIdx.y + j) > blockDim.y-1 )	// bottom boarder
          value = 0;
        else	// boundary-in case
          value = fid[pixel_address + i + j * M];
      }
    sum += value * h[smooth_L + i][smooth_L + j];
    }
smooth_fid[pixel_address] = sum; 
}

__global__ void edge(float *edge_fid,float *smooth_fid, int M)
{
const int edge_L = 3;
const int h_size = 7;
float edge_hx[h_size][h_size] = {-1, 0, 1, -2, 0, 2, -1,0,1}, edge_hy[h_size][h_size] = {-1, -2, -1, 0, 0, 0, 1,2,1};
const int pixel_address = threadIdx.x + (blockIdx.x *blockDim.x) +(threadIdx.y * M) +(blockIdx.y * blockDim.y) * M;
float sum1 = 0;
float sum2 = 0;
float value1 = 0;
float value2 = 0;

for (int i = -edge_L; i <= edge_L; i++)	
  for (int j = -edge_L; j <= edge_L; j++)	
    {
      // check row 
      if (blockIdx.x == 0 && (threadIdx.x + i) < 0)	// left boarder
        value1 = 0;
      else if ( blockIdx.x == (gridDim.x - 1) && (threadIdx.x + i) > blockDim.x-1 )	// right boarder
        value1 = 0;
      else 
      { 
        // check column 
        if (blockIdx.y == 0 && (threadIdx.y + j) < 0)	// top boarder
          value1 = 0;
        else if ( blockIdx.y == (gridDim.y - 1) && (threadIdx.y + j) > blockDim.y-1 )	// bottom boarder
          value1 = 0;
        else	// boundary-in case
          value1 = smooth_fid[pixel_address + i + j * M];
      } 
    sum1 += value1 * edge_hx[edge_L + i][edge_L + j];
    }
for (int i = -edge_L; i <= edge_L; i++)	
  for (int j = -edge_L; j <= edge_L; j++)	
    {
      // check row 
      if (blockIdx.x == 0 && (threadIdx.x + i) < 0)	// left boarder
        value2 = 0;
      else if ( blockIdx.x == (gridDim.x - 1) && (threadIdx.x + i) > blockDim.x-1 )	// right boarder
        value2 = 0;
      else 
      { 
        // check col 
        if (blockIdx.y == 0 && (threadIdx.y + j) < 0)	// top boarder
          value2 = 0;
        else if ( blockIdx.y == (gridDim.y - 1) && (threadIdx.y + j) > blockDim.y-1 )	// bottom boarder
          value2 = 0;
        else	// boundary-in case
          value2 = smooth_fid[pixel_address + i + j * M];
      } 
    sum2 += value2 * edge_hy[edge_L + i][edge_L + j];
    }    
    
edge_fid[pixel_address] = sqrt(((sum1*sum1) + (sum2*sum2))); 
}
main() {
/* Time Declaration */
struct timeb tmb;
struct timeb tma;
/* File Read Declaration*/
FILE *fr = fopen("Leaves_noise.bin", "r");
FILE *fw1 = fopen("1_Low_Pass_Filtered.bin", "w");
FILE *fw2 = fopen("2_Edge_Detected.bin", "w");
/* Declaration */
hipError_t er1,er2,er3,er4,er5;
const int smooth_W = 15;
const int edge_W = 7;
const int smooth_L = 7;
const int edge_L = 3;
const int segma = 1.5;
const int M = 2048;
const int B = 2;
const int image_size = M * M;
  /* GPU */
float* fid;  //float image for gpu
float* smooth_fid;  //Smoothed float image for gpu
float* edge_fid;  //Edged float image for gpu
  /* CPU */
float* fih;  //float image for cpu
float* smooth_fih;  //Smoothed float image for cpu
float* edge_fih;  //Edged float image for cpu
unsigned char* cih;  //unsighed char image for cpu
  /* Filter */
float* smooth_filter_d; //gpu smooth filter
float* smooth_filter_h; //cpu smooth filter

/* cuda memory allocation global*/
er1 = hipMalloc((void**) &fid, image_size * sizeof(float));
er2 = hipMalloc((void**) &smooth_fid, image_size * sizeof(float));
er3 = hipMalloc((void**) &edge_fid, image_size * sizeof(float));
er4 = hipMalloc((void**) &smooth_filter_d, smooth_W * smooth_W * sizeof(float));
if(er1 != 0 || er2 != 0 || er3 != 0 || er4 != 0) {
  printf("Cuda Success Failed\n");
  return 0;
}

/* CPU  Matrices Initilization*/
fih = (float*)malloc(image_size * sizeof(float));
smooth_fih = (float*)malloc(image_size * sizeof(float));
edge_fih = (float*)malloc(image_size * sizeof(float));
smooth_filter_h = (float*)malloc(smooth_W * smooth_W * sizeof(float));
cih = (unsigned char*)malloc(image_size * sizeof(unsigned char));
for (int y=0; y<M; y++) {
	for (int x=0; x<M; x++){
     if(y < smooth_W && x < smooth_W) {
       smooth_filter_h[y*smooth_W + x] = (1/(2*M_PI*pow(segma,2))) * exp(-((pow((y-smooth_L),2) + pow((x-smooth_L),2)) / (2*pow(segma,2))));
     }
     smooth_fih[y*M + x] = 0;
     edge_fih[y*M + x] = 0;
     fih[y*M + x] = 0;
     cih[y*M + x] = 0; 
	}
}

/* Orinigal Image Read*/
fread(cih, 1, M*M, fr);
fclose(fr);

/* Upcast Image to Float */
for (int y=0; y<M; y++) {
	for (int x=0; x<M; x++){
     fih[y*M + x] = cih[y*M + x]; 
	}
}
/* cuda setup dims */
dim3 dimBlock(B, B);
dim3 dimGrid(M/B, M/B);
/* Start Time */
ftime(&tmb);
/* cuda memory copy*/
hipMemcpy(fid, fih, image_size * sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(smooth_fid, smooth_fih, image_size * sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(edge_fid, edge_fih, image_size * sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(smooth_filter_d, smooth_filter_h, smooth_W * smooth_W * sizeof(float) , hipMemcpyHostToDevice);
/* launch cuda matrix multiplication*/
smooth<<<dimGrid, dimBlock>>>(smooth_fid, fid, smooth_filter_d,M);
edge<<<dimGrid, dimBlock>>>(edge_fid, smooth_fid,M);
/* CPU gets result from GPU*/
hipMemcpy(smooth_fih, smooth_fid, image_size * sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(edge_fih, edge_fid, image_size * sizeof(float), hipMemcpyDeviceToHost);
/* End Time */
ftime(&tma); 
printf("B = %d\n",B);
if((tma.millitm - tmb.millitm) < 0) {
  printf("Latency: %ld (seconds), ", tma.time - 1 - tmb.time);
  printf("%d (mlliseconds)\n", (tma.millitm + 1) - tmb.millitm);
}
else {
  printf("Latency: %ld (seconds), ", tma.time-tmb.time);
  printf("%d (mlliseconds)\n", tma.millitm - tmb.millitm);
}
/* Write Image Files */
  /* Downcast to unsigned char*/
for (int y=0; y<M; y++) {
	for (int x=0; x<M; x++){
    cih[y*M + x] = smooth_fih[y*M + x];
	}
}
  /* Smooth Image Write*/
if(fw1 != NULL) {
	fwrite(cih, 1, M*M, fw1);
}
else{
	printf("fw1: NULL\n");
}
fclose(fw1);

  /* Downcast to unsigned char*/
for (int y=0; y<M; y++) {
	for (int x=0; x<M; x++){        
    if((char)edge_fih[y*M + x] > 58) {
      cih[y*M + x] = 255;
    }
    else {
      cih[y*M + x] = 1;
    }
  }
}
  /* Edge Image Write*/
if(fw2!= NULL) {
	fwrite(cih, 1, M*M, fw2);
}
else{
	printf("fw2: NULL\n");
}
fclose(fw2);

/* Free allocations*/
hipFree(fid); hipFree(smooth_fid); hipFree(edge_fid); hipFree(smooth_filter_d);
free(fih); free(cih); free(smooth_fih); free(edge_fih); free(smooth_filter_h);
return 0;
}
